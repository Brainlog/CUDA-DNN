
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
#include <chrono>

#define KERNEL_SIZE 25

void softmax(float *vector, float *final_vector, int size)
{
    float denom = 0;
    float max = vector[0];
    for (int i = 1; i < size; i++)
    {
        if (vector[i] > max)
            max = vector[i];
    }
    for (int i = 0; i < size; i++)
        vector[i] -= max;
    for (int i = 0; i < size; i++)
        denom += exp(vector[i]);
    for (int i = 0; i < size; i++)
        final_vector[i] = exp(vector[i]) / denom;
}

__global__ void conv_kernel_p1(float *inp, float *out, int insize, float *kernel, int ksize, int inchannels, int kchannels, float *bias, int flag)
{
    int inchannel = blockIdx.y;
    int kchannel = blockIdx.x;
    int outchannel = (kchannel)*inchannels + inchannel;
    int row = threadIdx.x;
    int col = threadIdx.y;
    int outsize = insize - ksize + 1;
    float sum = 0;

    __shared__ float shared_kernel[KERNEL_SIZE];

    if(row == 0 && col == 0){
        for(int i = 0; i < ksize; i++)
        {
            for(int j = 0; j < ksize; j++)
            {
                shared_kernel[i*ksize + j] = kernel[outchannel * ksize * ksize + i * ksize + j];
            }
        }
    }

    __syncthreads();

    if (inchannel < inchannels && kchannel < kchannels && row < outsize && col < outsize)
    {
        for (int i = 0; i < ksize; i++)
        {
            for (int j = 0; j < ksize; j++)
            {
                sum += inp[inchannel * insize * insize + (row + i) * insize + col + j] * shared_kernel[i * ksize + j];
            }
        }
        if (flag == 0)
            out[outchannel  + (row * outsize + col)*(kchannels*inchannels)] = sum;
        else
            out[outchannel * outsize * outsize + row * outsize + col] = sum + bias[outchannel];
    }
}

__global__ void conv_kernel_p2(float *inp, float *out, int kchannels, int inchannels, int insize, float *bias)
{
    int kchannel = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    float temp = 0;
    if (kchannel < kchannels && row < insize && col < insize)
    {
        for (int i = 0; i < inchannels; i++)
        {
            int currchannel = inchannels * kchannel + i;
            temp += inp[currchannel  + (row * insize + col)*(kchannels*inchannels)];
        }
        out[kchannel * insize * insize + row * insize + col] = temp + bias[kchannel];
    }
}

__global__ void maxpool_kernel(float *inp, float *out, int insize, int ksize, int stride, int inchannels)
{
    int outsize = (insize - ksize) / stride + 1;
    int inchannel = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    int outchannel = inchannel;
    float maxval = 0;
    if (inchannel < inchannels && row % stride == 0 && col % stride == 0 && row + ksize - 1 < insize && col + ksize - 1 < insize)
    {
        int newrow = row / stride;
        int newcol = col / stride;
        for (int i = 0; i < ksize; i++)
        {
            for (int j = 0; j < ksize; j++)
            {
                maxval = max(maxval, inp[inchannel * insize * insize + (row + i) * insize + col + j]);
            }
        }
        out[outchannel * outsize * outsize + newrow * outsize + newcol] = maxval;
    }
}

__global__ void fc_kernel(float *inp, float *out, float *weight, float *bias, int insize, int outsize)
{
    int row = threadIdx.x;
    float sum = 0;
    for (int i = 0; i < insize; i++)
    {
        sum += inp[i] * weight[row * insize + i];
    }
    out[row] = sum + bias[row];
}

int main()
{
    // File extraction
    ofstream logger("./log.txt");
    ifstream conv1;
    conv1.open("./trained_weights/conv1.txt");
    ifstream conv2;
    conv2.open("./trained_weights/conv2.txt");
    ifstream conv3;
    conv3.open("./trained_weights/fc1.txt");
    ifstream fc2;
    fc2.open("./trained_weights/fc2.txt");

    // Conv1
    // Total filters = 20 Kernel size = 5 Input channels = 1 Output channels = 20 Input size = 28 Output size = 24 Bias = 20
    float *conv1_kernel = new float[20 * 5 * 5];
    float *conv1_bias = new float[20];

    // Reading weights and biases
    for (int i = 0; i < 20 * 5 * 5; i++)
    {
        conv1 >> conv1_kernel[i];
    }
    for (int i = 0; i < 20; i++)
    {
        conv1 >> conv1_bias[i];
    }

    // Conv2
    // assumed that filter[i][j] is the jth filter of the ith output channel
    // Total filters = 50x20, Kernel size = 5, Input channels = 20, Output channels = 50, Input size = 24, Output size = 8, Bias = 50
    float *conv2_kernel = new float[50 * 20 * 5 * 5];
    float *conv2_bias = new float[50];

    // Reading weights and biases
    for (int i = 0; i < 50 * 20 * 5 * 5; i++)
    {
        conv2 >> conv2_kernel[i];
    }
    for (int i = 0; i < 20; i++)
    {
        conv2 >> conv2_bias[i];
    }

    // Conv3
    // Total filters = 500, Kernel size = 4, Input channels = 50, Output channels = 500, Input size = 4, Output size = 1, Bias = 500
    float *conv3_kernel = new float[500 * 50 * 4 * 4];
    float *conv3_bias = new float[500];

    // Reading weights and biases
    for (int i = 0; i < 500 * 50 * 4 * 4; i++)
    {
        conv3 >> conv3_kernel[i];
    }
    for (int i = 0; i < 500; i++)
    {
        conv3 >> conv3_bias[i];
    }

    // FC2
    // Total weights = 10x500, Input size = 500, Output size = 10, Bias = 10
    float *fc2_weight = new float[10 * 500];
    float *fc2_bias = new float[10];

    // Reading weights and biases
    for (int i = 0; i < 10 * 500; i++)
    {
        fc2 >> fc2_weight[i];
    }
    for (int i = 0; i < 10; i++)
    {
        fc2 >> fc2_bias[i];
    }

    // input dataset and labels
    ifstream dataset;
    dataset.open("./test_dataset.txt");
    ifstream labels;
    labels.open("./test_labels.txt");
    float *inpu = new float[10000 * 28 * 28];
    int *label = new int[10000];
    for (int i = 0; i < 10000; i++)
    {
        labels >> label[i];
        for (int j = 0; j < 28 * 28; j++)
        {
            dataset >> inpu[i * 28 * 28 + j];
        }
    }
    // Device memory allocation for weights and biases
    float *d_conv1_kernel;
    float *d_conv1_bias;
    float *d_conv2_kernel;
    float *d_conv2_bias;
    float *d_conv3_kernel;
    float *d_conv3_bias;
    float *d_fc2_weight;
    float *d_fc2_bias;

    hipMalloc(&d_conv1_kernel, 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv1_bias, 20 * sizeof(float));
    hipMalloc(&d_conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv2_bias, 50 * sizeof(float));
    hipMalloc(&d_conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float));
    hipMalloc(&d_conv3_bias, 500 * sizeof(float));
    hipMalloc(&d_fc2_weight, 10 * 500 * sizeof(float));
    hipMalloc(&d_fc2_bias, 10 * sizeof(float));
    hipMemcpy(d_conv1_kernel, conv1_kernel, 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_bias, 20 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_kernel, conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_bias, 50 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_kernel, conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_bias, conv3_bias, 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_weight, fc2_weight, 10 * 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_bias, fc2_bias, 10 * sizeof(float), hipMemcpyHostToDevice);

    // Device memory allocation for input and output
    int batch = 10000;
    int start = 0;
    int count = 0;
    float *inp = new float[28 * 28];
    float *d_inp;
    hipMalloc(&d_inp, 28 * 28 * sizeof(float));
    float *d_out1_p1;
    float *d_out2;
    float *d_out3_p1;
    float *d_out3_p2;
    float *d_out4;
    float *d_out5_p1;
    float *d_out5_p2;
    float *d_out6;
    hipMalloc(&d_out1_p1, 20 * 24 * 24 * sizeof(float));
    hipMalloc(&d_out2, 20 * 12 * 12 * sizeof(float));
    hipMalloc(&d_out3_p1, 20 * 50 * 8 * 8 * sizeof(float));
    hipMalloc(&d_out3_p2, 50 * 8 * 8 * sizeof(float));
    hipMalloc(&d_out4, 50 * 4 * 4 * sizeof(float));
    hipMalloc(&d_out5_p1, 50 * 500 * sizeof(float));
    hipMalloc(&d_out5_p2, 500 * sizeof(float));
    hipMalloc(&d_out6, 10 * sizeof(float));
    auto str = std::chrono::high_resolution_clock::now();
    // count time
    for (int i = start; i < start + batch; i++)
    {
        // Dimensions of all outputs :
        // Conv1 : 20x24x24, Pool1 : 20x12x12, Conv2 : 50x8x8, Pool2 : 50x4x4, Conv3 : 500, FC2 : 10

        for (int j = 0; j < 28 * 28; j++)
        {
            inp[j] = inpu[i * 28 * 28 + j];
        }
        hipMemcpy(d_inp, inp, 28 * 28 * sizeof(float), hipMemcpyHostToDevice);

        // Perform Inference
        // Conv1
        int ksize = 5;
        int insize = 28;
        int inchannels = 1;
        int kchannels = 20;
        dim3 threads1(24, 24);
        dim3 blocks1(20,1);
        int flag = 1;
        conv_kernel_p1<<<blocks1, threads1>>>(d_inp, d_out1_p1, insize, d_conv1_kernel, ksize, inchannels, kchannels, d_conv1_bias, flag); 

        // Pool1
        ksize = 2;
        int stride = 2;
        insize = 24;
        inchannels = 20;
        dim3 threads2(24, 24);
        dim3 blocks2(20);
        maxpool_kernel<<<blocks2, threads2>>>(d_out1_p1, d_out2, insize, ksize, stride, inchannels);


        // Conv2
        ksize = 5;
        insize = 12;
        inchannels = 20;
        kchannels = 50;
        dim3 threads3(8, 8);
        dim3 threads3_2(8,8);
        dim3 blocks3(50,20);
        conv_kernel_p1<<<blocks3, threads3>>>(d_out2, d_out3_p1, insize, d_conv2_kernel, ksize, inchannels, kchannels, d_conv2_bias, 0);
        conv_kernel_p2<<<50, threads3_2>>>(d_out3_p1, d_out3_p2, kchannels, inchannels, insize - ksize + 1, d_conv2_bias);


        // Pool2
        ksize = 2;
        stride = 2;
        insize = 8;
        inchannels = 50;
        dim3 threads4(8, 8);
        dim3 blocks4(50);
        maxpool_kernel<<<blocks4, threads4>>>(d_out3_p2, d_out4, insize, ksize, stride, inchannels);
        hipMemset(d_out3_p2, 0, 50 * 8 * 8 * sizeof(float));
  

        // Conv3
        ksize = 4;
        insize = 4;
        inchannels = 50;
        kchannels = 500;
        dim3 threads5(1, 1);
        dim3 threads5_2(1,1);
        dim3 blocks5(500, 50);
        conv_kernel_p1<<<blocks5, threads5>>>(d_out4, d_out5_p1, insize, d_conv3_kernel, ksize, inchannels, kchannels, d_conv3_bias, 0);
        conv_kernel_p2<<<500, threads5_2>>>(d_out5_p1, d_out5_p2, kchannels, inchannels, insize - ksize + 1, d_conv3_bias);


        // FC2
        insize = 500;
        int outsize = 10;
        dim3 threads6(10);
        fc_kernel<<<1, threads6>>>(d_out5_p2, d_out6, d_fc2_weight, d_fc2_bias, insize, outsize);
        hipMemset(d_out5_p2, 0, 500 * sizeof(float));
        // Probabilities
        float *out6 = new float[10];
        hipMemcpy(out6, d_out6, 10 * sizeof(float), hipMemcpyDeviceToHost);

        float *final_out = new float[10];
        softmax(out6, final_out, 10);
        int max_index = 0;
        for (int j = 0; j < 10; j++)
        {
            if (final_out[j] > final_out[max_index])
            {
                max_index = j;
            }
        }

        if (label[i] == max_index)
        {
            count++;
        }

    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - str);
    std::cout << "Total Time : " << duration.count() << "\n";
    std::cout << "Accuracy : " << count << " / " << batch << endl;

    hipFree(d_inp);
    hipFree(d_out1_p1);
    hipFree(d_out2);
    hipFree(d_out3_p1);
    hipFree(d_out3_p2);
    hipFree(d_out4);
    hipFree(d_out5_p1);
    hipFree(d_out5_p2);
    hipFree(d_out6);

    // Free the memory of weights
    hipFree(d_conv1_kernel);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_kernel);
    hipFree(d_conv2_bias);
    hipFree(d_conv3_kernel);
    hipFree(d_conv3_bias);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);

    // Free host memory
    delete[] inpu;
    delete[] label;
    delete[] conv1_kernel;
    delete[] conv1_bias;
    delete[] conv2_kernel;
    delete[] conv2_bias;
    delete[] conv3_kernel;
    delete[] conv3_bias;
    delete[] fc2_weight;
    delete[] fc2_bias;

    return 0;
}
