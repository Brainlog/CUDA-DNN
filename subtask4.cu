
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
#include <chrono>

#define SHARED_FC_INPUT_SIZE 500

void softmax(float *vector, float *final_vector, int size)
{
    float denom = 0;
    float max = vector[0];
    for (int i = 1; i < size; i++)
    {
        if (vector[i] > max)
            max = vector[i];
    }
    for (int i = 0; i < size; i++)
        vector[i] -= max;
    for (int i = 0; i < size; i++)
        denom += exp(vector[i]);
    for (int i = 0; i < size; i++)
        final_vector[i] = exp(vector[i]) / denom;
}

__global__ void conv_kernel_p1(float *inp, float *out, int insize, float *kernel, int ksize, int inchannels, int kchannels, float *bias, int flag)
{
    int inchannel = blockIdx.y;
    int kchannel = blockIdx.x;
    int outchannel = (kchannel)*inchannels + inchannel;
    int row = threadIdx.x;
    int col = threadIdx.y;
    int outsize = insize - ksize + 1;
    float sum = 0;
    if (inchannel < inchannels && kchannel < kchannels && row < outsize && col < outsize)
    {
        for (int i = 0; i < ksize; i++)
        {
            for (int j = 0; j < ksize; j++)
            {
                sum += inp[inchannel * insize * insize + (row + i) * insize + col + j] * kernel[outchannel * ksize * ksize + i * ksize + j];
            }
        }
        if (flag == 0)
            out[outchannel + (row * outsize + col) * (kchannels * inchannels)] = sum;
        else
            out[outchannel * outsize * outsize + row * outsize + col] = sum + bias[outchannel];
    }
}

__global__ void conv_kernel_p2(float *inp, float *out, int kchannels, int inchannels, int insize, float *bias)
{
    int kchannel = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    float temp = 0;
    if (kchannel < kchannels && row < insize && col < insize)
    {
        for (int i = 0; i < inchannels; i++)
        {
            int currchannel = inchannels * kchannel + i;
            temp += inp[currchannel + (row * insize + col) * (kchannels * inchannels)];
        }
        out[kchannel * insize * insize + row * insize + col] = temp + bias[kchannel];
    }
}

__global__ void maxpool_kernel(float *inp, float *out, int insize, int ksize, int stride, int inchannels)
{
    int outsize = (insize - ksize) / stride + 1;
    int inchannel = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    int outchannel = inchannel;
    float maxval = -1000;
    if (inchannel < inchannels && row % stride == 0 && col % stride == 0 && row + ksize - 1 < insize && col + ksize - 1 < insize)
    {
        int newrow = row / stride;
        int newcol = col / stride;
        for (int i = 0; i < ksize; i++)
        {
            for (int j = 0; j < ksize; j++)
            {
                maxval = max(maxval, inp[inchannel * insize * insize + (row + i) * insize + col + j]);
            }
        }
        out[outchannel * outsize * outsize + newrow * outsize + newcol] = maxval;
    }
}

__global__ void fc_kernel(float *inp, float *out, float *weight, float *bias, int insize, int outsize)
{
    int row = threadIdx.x;
    float sum = 0;

    __shared__ float shared_fc_input[SHARED_FC_INPUT_SIZE];

    for (int i = 0; i < 50; i++){
        shared_fc_input[row * 50 + i] = inp[row * 50 + i];
        if(shared_fc_input[row * 50 + i] < 0)
            shared_fc_input[row * 50 + i] = 0;
    }

    __syncthreads();

    for (int i = 0; i < insize; i++)
    {
        sum += shared_fc_input[i] * weight[row * insize + i];
    }
    out[row] = sum + bias[row];
}

void process_with_streams()
{
    // File extraction
    ofstream logger("./log.txt");
    ifstream conv1;
    conv1.open("./trained_weights/conv1.txt");
    ifstream conv2;
    conv2.open("./trained_weights/conv2.txt");
    ifstream conv3;
    conv3.open("./trained_weights/fc1.txt");
    ifstream fc2;
    fc2.open("./trained_weights/fc2.txt");

    // Conv1
    // Total filters = 20 Kernel size = 5 Input channels = 1 Output channels = 20 Input size = 28 Output size = 24 Bias = 20
    float *conv1_kernel = new float[20 * 5 * 5];
    float *conv1_bias = new float[20];

    // Reading weights and biases
    for (int i = 0; i < 20 * 5 * 5; i++)
    {
        conv1 >> conv1_kernel[i];
    }
    for (int i = 0; i < 20; i++)
    {
        conv1 >> conv1_bias[i];
    }

    // Conv2
    // assumed that filter[i][j] is the jth filter of the ith output channel
    // Total filters = 50x20, Kernel size = 5, Input channels = 20, Output channels = 50, Input size = 24, Output size = 8, Bias = 50
    float *conv2_kernel = new float[50 * 20 * 5 * 5];
    float *conv2_bias = new float[50];

    // Reading weights and biases
    for (int i = 0; i < 50 * 20 * 5 * 5; i++)
    {
        conv2 >> conv2_kernel[i];
    }
    for (int i = 0; i < 50; i++)
    {
        conv2 >> conv2_bias[i];
    }

    // Conv3
    // Total filters = 500, Kernel size = 4, Input channels = 50, Output channels = 500, Input size = 4, Output size = 1, Bias = 500
    float *conv3_kernel = new float[500 * 50 * 4 * 4];
    float *conv3_bias = new float[500];

    // Reading weights and biases
    for (int i = 0; i < 500 * 50 * 4 * 4; i++)
    {
        conv3 >> conv3_kernel[i];
    }
    for (int i = 0; i < 500; i++)
    {
        conv3 >> conv3_bias[i];
    }

    // FC2
    // Total weights = 10x500, Input size = 500, Output size = 10, Bias = 10
    float *fc2_weight = new float[10 * 500];
    float *fc2_bias = new float[10];

    // Reading weights and biases
    for (int i = 0; i < 10 * 500; i++)
    {
        fc2 >> fc2_weight[i];
    }
    for (int i = 0; i < 10; i++)
    {
        fc2 >> fc2_bias[i];
    }

    // input dataset and labels
    ifstream dataset;
    dataset.open("./test_dataset.txt");
    ifstream labels;
    labels.open("./test_labels.txt");
    float *inpu = new float[10000 * 28 * 28];
    int *label = new int[10000];
    for (int i = 0; i < 10000; i++)
    {
        labels >> label[i];
        for (int j = 0; j < 28 * 28; j++)
        {
            dataset >> inpu[i * 28 * 28 + j];
        }
    }
    // Device memory allocation for weights and biases
    float *d_conv1_kernel;
    float *d_conv1_bias;
    float *d_conv2_kernel;
    float *d_conv2_bias;
    float *d_conv3_kernel;
    float *d_conv3_bias;
    float *d_fc2_weight;
    float *d_fc2_bias;

    hipMalloc(&d_conv1_kernel, 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv1_bias, 20 * sizeof(float));
    hipMalloc(&d_conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv2_bias, 50 * sizeof(float));
    hipMalloc(&d_conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float));
    hipMalloc(&d_conv3_bias, 500 * sizeof(float));
    hipMalloc(&d_fc2_weight, 10 * 500 * sizeof(float));
    hipMalloc(&d_fc2_bias, 10 * sizeof(float));
    hipMemcpy(d_conv1_kernel, conv1_kernel, 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_bias, 20 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_kernel, conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_bias, 50 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_kernel, conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_bias, conv3_bias, 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_weight, fc2_weight, 10 * 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_bias, fc2_bias, 10 * sizeof(float), hipMemcpyHostToDevice);

    // Inference details
    int batch = 10000;
    int num_sub_batches = 100;
    int sub_batch_size = batch / num_sub_batches;

    int count = 0;
    int ksize1 = 5;
    int insize1 = 28;
    int inchannels1 = 1;
    int kchannels1 = 20;
    dim3 threads1(24, 24);
    dim3 blocks1(20, 1);
    int flag1 = 1;
    int ksize2 = 2;
    int stride2 = 2;
    int insize2 = 24;
    int inchannels2 = 20;
    dim3 threads2(24, 24);
    dim3 blocks2(20);
    int ksize3 = 5;
    int insize3 = 12;
    int inchannels3 = 20;
    int kchannels3 = 50;
    dim3 threads3(8, 8);
    dim3 threads3_2(8, 8);
    dim3 blocks3(50, 20);
    int ksize4 = 2;
    int stride4 = 2;
    int insize4 = 8;
    int inchannels4 = 50;
    dim3 threads4(8, 8);
    dim3 blocks4(50);
    int ksize5 = 4;
    int insize5 = 4;
    int inchannels5 = 50;
    int kchannels5 = 500;
    dim3 threads5(1, 1);
    dim3 threads5_2(1, 1);
    dim3 blocks5(500, 50);
    int insize6 = 500;
    int outsize6 = 10;
    dim3 threads6(10);

    float **out_probs = new float *[batch];
    for(int i = 0; i < batch; i++){
        out_probs[i] = new float[10];
    }

    // Memory allocation for input and output
    // float *inp = new float[28 * 28];
    float **inp = new float *[batch];
    for(int i = 0; i < batch; i++){
        inp[i] = new float[28 * 28];
    }
    float **d_inp = new float *[num_sub_batches];
    float **d_out1_p1 = new float *[num_sub_batches];
    float **d_out2 = new float *[num_sub_batches];
    float **d_out3_p1 = new float *[num_sub_batches];
    float **d_out3_p2 = new float *[num_sub_batches];
    float **d_out4 = new float *[num_sub_batches];
    float **d_out5_p1 = new float *[num_sub_batches];
    float **d_out5_p2 = new float *[num_sub_batches];
    float **d_out6 = new float *[num_sub_batches];
    for (int i = 0; i < num_sub_batches; i++)
    {
        hipMalloc(&d_inp[i], 28 * 28 * sizeof(float));
        hipMalloc(&d_out1_p1[i], 20 * 24 * 24 * sizeof(float));
        hipMalloc(&d_out2[i], 20 * 12 * 12 * sizeof(float));
        hipMalloc(&d_out3_p1[i], 20 * 50 * 8 * 8 * sizeof(float));
        hipMalloc(&d_out3_p2[i], 50 * 8 * 8 * sizeof(float));
        hipMalloc(&d_out4[i], 50 * 4 * 4 * sizeof(float));
        hipMalloc(&d_out5_p1[i], 50 * 500 * sizeof(float));
        hipMalloc(&d_out5_p2[i], 500 * sizeof(float));
        hipMalloc(&d_out6[i], 10 * sizeof(float));
    }
    hipDeviceSynchronize();
    for(int i = 0; i < batch; i++){
        for(int j = 0; j < 28 * 28; j++){
            inp[i][j] = inpu[i * 28 * 28 + j];
        }
    }

    auto str = std::chrono::high_resolution_clock::now();

    hipStream_t* streams = new hipStream_t[num_sub_batches];
    for(int i = 0; i < num_sub_batches; i++){
        hipStreamCreate(&streams[i]);
    }

    for(int j = 0; j < sub_batch_size; j++){
        for(int i = 0; i < num_sub_batches; i++){
            hipMemcpyAsync(d_inp[i], inp[j * num_sub_batches + i], 28 * 28 * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        }
        for(int i = 0; i < num_sub_batches; i++){
            conv_kernel_p1<<<blocks1, threads1, 0, streams[i]>>>(d_inp[i], d_out1_p1[i], insize1, d_conv1_kernel, ksize1, inchannels1, kchannels1, d_conv1_bias, flag1);
        }
        for(int i = 0; i < num_sub_batches; i++){
            maxpool_kernel<<<blocks2, threads2, 0, streams[i]>>>(d_out1_p1[i], d_out2[i], insize2, ksize2, stride2, inchannels2);
        }
        for(int i = 0; i < num_sub_batches; i++){
            conv_kernel_p1<<<blocks3, threads3, 0, streams[i]>>>(d_out2[i], d_out3_p1[i], insize3, d_conv2_kernel, ksize3, inchannels3, kchannels3, d_conv2_bias, 0);
        }
        for(int i = 0; i < num_sub_batches; i++){
            conv_kernel_p2<<<50, threads3_2, 0, streams[i]>>>(d_out3_p1[i], d_out3_p2[i], kchannels3, inchannels3, insize3 - ksize3 + 1, d_conv2_bias);
        }
        for (int i = 0; i < num_sub_batches; i++)
            maxpool_kernel<<<blocks4, threads4, 0 ,streams[i]>>>(d_out3_p2[i], d_out4[i], insize4, ksize4, stride4, inchannels4);
        for (int i = 0; i < num_sub_batches; i++)
            conv_kernel_p1<<<blocks5, threads5, 0, streams[i]>>>(d_out4[i], d_out5_p1[i], insize5, d_conv3_kernel, ksize5, inchannels5, kchannels5, d_conv3_bias, 0);
        for (int i = 0; i < num_sub_batches; i++)
            conv_kernel_p2<<<500, threads5_2, 0, streams[i]>>>(d_out5_p1[i], d_out5_p2[i], kchannels5, inchannels5, insize5 - ksize5 + 1, d_conv3_bias);

            // FC2
        for (int i = 0; i < num_sub_batches; i++)
            fc_kernel<<<1, threads6, 0, streams[i]>>>(d_out5_p2[i], d_out6[i], d_fc2_weight, d_fc2_bias, insize6, outsize6);

        // // Probabilities
        float** out6 = new float*[num_sub_batches];
        for(int i = 0; i < num_sub_batches; i++){
            out6[i] = new float[10];
            hipMemcpy(out6[i], d_out6[i], 10 * sizeof(float), hipMemcpyDeviceToHost);
            // float *final_out = new float[10];
            softmax(out6[i], out_probs[j * num_sub_batches + i], 10);
            int max_index = 0;
            for (int k = 0; k < 10; k++)
            {
                if (out_probs[j * num_sub_batches + i][k] > out_probs[j * num_sub_batches + i][max_index])
                {
                    max_index = k;
                }
            }

            if(label[j * num_sub_batches + i] == max_index){
                count++;
            }
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - str);

    ofstream out_file;
    out_file.open("output_subtask_4.txt");

    for(int i=0; i<batch; i++){
        sort(out_probs[i], out_probs[i] + 10, greater<float>());
        out_file << "Image " << i << " : " << out_probs[i][0] << ", " << out_probs[i][1] << ", " << out_probs[i][2] << ", " << out_probs[i][3] << ", " << out_probs[i][4] << endl;
    }

    out_file.close();

    for(int i = 0; i < batch; i++){
        delete[] out_probs[i];
    }

    std::cout << "Total Time : " << duration.count() << "\n";
    std::cout << "Accuracy : " << count << " / " << batch << endl;

    for(int i =0; i < num_sub_batches; i++){
        hipStreamDestroy(streams[i]);
    }

    hipFree(d_inp);
    hipFree(d_out1_p1);
    hipFree(d_out2);
    hipFree(d_out3_p1);
    hipFree(d_out3_p2);
    hipFree(d_out4);
    hipFree(d_out5_p1);
    hipFree(d_out5_p2);
    hipFree(d_out6);

    // Free the memory of weights
    hipFree(d_conv1_kernel);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_kernel);
    hipFree(d_conv2_bias);
    hipFree(d_conv3_kernel);
    hipFree(d_conv3_bias);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);

    // Free host memory
    delete[] inpu;
    delete[] label;
    delete[] conv1_kernel;
    delete[] conv1_bias;
    delete[] conv2_kernel;
    delete[] conv2_bias;
    delete[] conv3_kernel;
    delete[] conv3_bias;
    delete[] fc2_weight;
    delete[] fc2_bias;

    return;
}

void process_wo_streams()
{
    // File extraction
    ofstream logger("./log.txt");
    ifstream conv1;
    conv1.open("./trained_weights/conv1.txt");
    ifstream conv2;
    conv2.open("./trained_weights/conv2.txt");
    ifstream conv3;
    conv3.open("./trained_weights/fc1.txt");
    ifstream fc2;
    fc2.open("./trained_weights/fc2.txt");

    // Conv1
    // Total filters = 20 Kernel size = 5 Input channels = 1 Output channels = 20 Input size = 28 Output size = 24 Bias = 20
    float *conv1_kernel = new float[20 * 5 * 5];
    float *conv1_bias = new float[20];

    // Reading weights and biases
    for (int i = 0; i < 20 * 5 * 5; i++)
    {
        conv1 >> conv1_kernel[i];
    }
    for (int i = 0; i < 20; i++)
    {
        conv1 >> conv1_bias[i];
    }

    // Conv2
    // assumed that filter[i][j] is the jth filter of the ith output channel
    // Total filters = 50x20, Kernel size = 5, Input channels = 20, Output channels = 50, Input size = 24, Output size = 8, Bias = 50
    float *conv2_kernel = new float[50 * 20 * 5 * 5];
    float *conv2_bias = new float[50];

    // Reading weights and biases
    for (int i = 0; i < 50 * 20 * 5 * 5; i++)
    {
        conv2 >> conv2_kernel[i];
    }
    for (int i = 0; i < 50; i++)
    {
        conv2 >> conv2_bias[i];
    }

    // Conv3
    // Total filters = 500, Kernel size = 4, Input channels = 50, Output channels = 500, Input size = 4, Output size = 1, Bias = 500
    float *conv3_kernel = new float[500 * 50 * 4 * 4];
    float *conv3_bias = new float[500];

    // Reading weights and biases
    for (int i = 0; i < 500 * 50 * 4 * 4; i++)
    {
        conv3 >> conv3_kernel[i];
    }
    for (int i = 0; i < 500; i++)
    {
        conv3 >> conv3_bias[i];
    }

    // FC2
    // Total weights = 10x500, Input size = 500, Output size = 10, Bias = 10
    float *fc2_weight = new float[10 * 500];
    float *fc2_bias = new float[10];

    // Reading weights and biases
    for (int i = 0; i < 10 * 500; i++)
    {
        fc2 >> fc2_weight[i];
    }
    for (int i = 0; i < 10; i++)
    {
        fc2 >> fc2_bias[i];
    }

    // input dataset and labels
    ifstream dataset;
    dataset.open("./test_dataset.txt");
    ifstream labels;
    labels.open("./test_labels.txt");
    float *inpu = new float[10000 * 28 * 28];
    int *label = new int[10000];
    for (int i = 0; i < 10000; i++)
    {
        labels >> label[i];
        for (int j = 0; j < 28 * 28; j++)
        {
            dataset >> inpu[i * 28 * 28 + j];
        }
    }
    // Device memory allocation for weights and biases
    float *d_conv1_kernel;
    float *d_conv1_bias;
    float *d_conv2_kernel;
    float *d_conv2_bias;
    float *d_conv3_kernel;
    float *d_conv3_bias;
    float *d_fc2_weight;
    float *d_fc2_bias;

    hipMalloc(&d_conv1_kernel, 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv1_bias, 20 * sizeof(float));
    hipMalloc(&d_conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float));
    hipMalloc(&d_conv2_bias, 50 * sizeof(float));
    hipMalloc(&d_conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float));
    hipMalloc(&d_conv3_bias, 500 * sizeof(float));
    hipMalloc(&d_fc2_weight, 10 * 500 * sizeof(float));
    hipMalloc(&d_fc2_bias, 10 * sizeof(float));
    hipMemcpy(d_conv1_kernel, conv1_kernel, 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv1_bias, conv1_bias, 20 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_kernel, conv2_kernel, 50 * 20 * 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv2_bias, conv2_bias, 50 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_kernel, conv3_kernel, 500 * 50 * 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_conv3_bias, conv3_bias, 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_weight, fc2_weight, 10 * 500 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_fc2_bias, fc2_bias, 10 * sizeof(float), hipMemcpyHostToDevice);

    // Device memory allocation for input and output
    int batch = 10000;
    int start = 0;
    int count = 0;
    float *inp = new float[28 * 28];
    float *d_inp;
    hipMalloc(&d_inp, 28 * 28 * sizeof(float));
    float *d_out1_p1;
    float *d_out2;
    float *d_out3_p1;
    float *d_out3_p2;
    float *d_out4;
    float *d_out5_p1;
    float *d_out5_p2;
    float *d_out6;
    hipMalloc(&d_out1_p1, 20 * 24 * 24 * sizeof(float));
    hipMalloc(&d_out2, 20 * 12 * 12 * sizeof(float));
    hipMalloc(&d_out3_p1, 20 * 50 * 8 * 8 * sizeof(float));
    hipMalloc(&d_out3_p2, 50 * 8 * 8 * sizeof(float));
    hipMalloc(&d_out4, 50 * 4 * 4 * sizeof(float));
    hipMalloc(&d_out5_p1, 50 * 500 * sizeof(float));
    hipMalloc(&d_out5_p2, 500 * sizeof(float));
    hipMalloc(&d_out6, 10 * sizeof(float));
    auto str = std::chrono::high_resolution_clock::now();
    
    float **out_probs = new float *[batch];
    for(int i = 0; i < batch; i++){
        out_probs[i] = new float[10];
    }
    // count time
    for (int i = start; i < start + batch; i++)
    {
        // Dimensions of all outputs :
        // Conv1 : 20x24x24, Pool1 : 20x12x12, Conv2 : 50x8x8, Pool2 : 50x4x4, Conv3 : 500, FC2 : 10

        for (int j = 0; j < 28 * 28; j++)
        {
            inp[j] = inpu[i * 28 * 28 + j];
        }
        hipMemcpy(d_inp, inp, 28 * 28 * sizeof(float), hipMemcpyHostToDevice);

        // Perform Inference
        // Conv1
        int ksize = 5;
        int insize = 28;
        int inchannels = 1;
        int kchannels = 20;
        dim3 threads1(24, 24);
        dim3 blocks1(20,1);
        int flag = 1;
        conv_kernel_p1<<<blocks1, threads1>>>(d_inp, d_out1_p1, insize, d_conv1_kernel, ksize, inchannels, kchannels, d_conv1_bias, flag); 

        // Pool1
        ksize = 2;
        int stride = 2;
        insize = 24;
        inchannels = 20;
        dim3 threads2(24, 24);
        dim3 blocks2(20);
        maxpool_kernel<<<blocks2, threads2>>>(d_out1_p1, d_out2, insize, ksize, stride, inchannels);


        // Conv2
        ksize = 5;
        insize = 12;
        inchannels = 20;
        kchannels = 50;
        dim3 threads3(8, 8);
        dim3 threads3_2(8,8);
        dim3 blocks3(50,20);
        conv_kernel_p1<<<blocks3, threads3>>>(d_out2, d_out3_p1, insize, d_conv2_kernel, ksize, inchannels, kchannels, d_conv2_bias, 0);
        conv_kernel_p2<<<50, threads3_2>>>(d_out3_p1, d_out3_p2, kchannels, inchannels, insize - ksize + 1, d_conv2_bias);


        // Pool2
        ksize = 2;
        stride = 2;
        insize = 8;
        inchannels = 50;
        dim3 threads4(8, 8);
        dim3 blocks4(50);
        maxpool_kernel<<<blocks4, threads4>>>(d_out3_p2, d_out4, insize, ksize, stride, inchannels);
        hipMemset(d_out3_p2, 0, 50 * 8 * 8 * sizeof(float));
  

        // Conv3
        ksize = 4;
        insize = 4;
        inchannels = 50;
        kchannels = 500;
        dim3 threads5(1, 1);
        dim3 threads5_2(1,1);
        dim3 blocks5(500, 50);
        conv_kernel_p1<<<blocks5, threads5>>>(d_out4, d_out5_p1, insize, d_conv3_kernel, ksize, inchannels, kchannels, d_conv3_bias, 0);
        conv_kernel_p2<<<500, threads5_2>>>(d_out5_p1, d_out5_p2, kchannels, inchannels, insize - ksize + 1, d_conv3_bias);


        // FC2
        insize = 500;
        int outsize = 10;
        dim3 threads6(10);
        fc_kernel<<<1, threads6>>>(d_out5_p2, d_out6, d_fc2_weight, d_fc2_bias, insize, outsize);
        hipMemset(d_out5_p2, 0, 500 * sizeof(float));

        // Probabilities
        float *out6 = new float[10];
        hipMemcpy(out6, d_out6, 10 * sizeof(float), hipMemcpyDeviceToHost);

        // float *final_out = new float[10];
        softmax(out6, out_probs[i], 10);
        int max_index = 0;
        
        for (int j = 0; j < 10; j++)
        {
            if (out_probs[i][j] > out_probs[i][max_index])
            {
                max_index = j;
            }
        }
        
        if (label[i] == max_index)
        {
            count++;
        }
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::milliseconds duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - str);

    ofstream out_file;
    out_file.open("output_subtask_4.txt");

    for(int i=0; i<batch; i++){
        sort(out_probs[i], out_probs[i] + 10, greater<float>());
        out_file << "Image " << i << " : " << out_probs[i][0] << ", " << out_probs[i][1] << ", " << out_probs[i][2] << ", " << out_probs[i][3] << ", " << out_probs[i][4] << endl;
    }

    out_file.close();

    for(int i = 0; i < batch; i++){
        delete[] out_probs[i];
    }

    std::cout << "Total Time : " << duration.count() << "\n";
    std::cout << "Accuracy : " << count << " / " << batch << endl;

    hipFree(d_inp);
    hipFree(d_out1_p1);
    hipFree(d_out2);
    hipFree(d_out3_p1);
    hipFree(d_out3_p2);
    hipFree(d_out4);
    hipFree(d_out5_p1);
    hipFree(d_out5_p2);
    hipFree(d_out6);

    // Free the memory of weights
    hipFree(d_conv1_kernel);
    hipFree(d_conv1_bias);
    hipFree(d_conv2_kernel);
    hipFree(d_conv2_bias);
    hipFree(d_conv3_kernel);
    hipFree(d_conv3_bias);
    hipFree(d_fc2_weight);
    hipFree(d_fc2_bias);

    // Free host memory
    delete[] inpu;
    delete[] label;
    delete[] conv1_kernel;
    delete[] conv1_bias;
    delete[] conv2_kernel;
    delete[] conv2_bias;
    delete[] conv3_kernel;
    delete[] conv3_bias;
    delete[] fc2_weight;
    delete[] fc2_bias;

    return;
}

int main(int argc, char **argv)
{
    if(argc < 2){
        cout << "Please provide input in the following format: 1 - with streams, 0 - without streams" << endl;
        return 0;
    }
    int mode = atoi(argv[1]);
    if(mode == 1){
        process_with_streams();
    }
    else{
        process_wo_streams();
    }
    return 0;
} 